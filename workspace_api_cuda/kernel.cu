
#include <hip/hip_runtime.h>
extern "C" __device__
float get_fraction(double from_value, double to_value, double level){
    if (to_value == from_value)
        return 0;
    return ((level - from_value) / (to_value - from_value));
}

extern "C" __global__
void saxpy(double *image, double *result_1x, double *result_1y, double *result_2x, double *result_2y, double level, size_t n, size_t width, size_t height)
{        
    size_t square_case;
    size_t r0 = blockIdx.y * blockDim.y + threadIdx.y;
    size_t c0 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t r1 = r0 + 1;
    size_t c1 = c0 + 1;

    struct tuple {
        size_t x;
        size_t y;
    } top, bottom, left, right;
    
    if( r0 < height && c0 < width ){

        // skip mask

        double ul = image[ r0 * width + c0 ];
        double ur = image[ r0 * width + c1 ];
        double ll = image[ r1 * width + c0 ];
        double lr = image[ r1 * width + c1 ];

        // skip control for NaN values

        square_case = 0;
        if (ul > level) square_case += 1;
        if (ur > level) square_case += 2;
        if (ll > level) square_case += 4;
        if (lr > level) square_case += 8; 

        if (square_case == 0 || square_case == 15){
            //TODO 
            result_1x[r0 * width + c0] = 0.0;
            result_1y[r0 * width + c0] = 0.0; 
            result_2x[r0 * width + c0] = 0.0;
            result_2y[r0 * width + c0] = 0.0;
        }

        top.x = r0; 
        top.y = c0 + get_fraction(ul,ur,level);
        bottom.x = r1;
        bottom.y = c0 + get_fraction(ll, lr, level);
        left.x = r0 + get_fraction(ul, ll, level);
        left.y = c0;
        right.x = r0 + get_fraction(ur, lr, level);
        right.y = c1;

        //result[r0*width+c0] = square_case;

        if (square_case == 1){
            result_1x[ r0 * width + c0 ] = top.x;
            result_1y[ r0 * width + c0 ] = top.y;
            result_2x[ r0 * width + c0 ] = left.x;
            result_2y[ r0 * width + c0 ] = left.y; 
        }
        else if (square_case == 2){
            result_1x[ r0 * width + c0 ] = right.x;
            result_1y[ r0 * width + c0 ] = right.y;
            result_2x[ r0 * width + c0 ] = top.x;
            result_2y[ r0 * width + c0 ] = top.y; 
        }
        else if (square_case == 3){
            result_1x[ r0 * width + c0 ] = right.x;
            result_1y[ r0 * width + c0 ] = right.y;
            result_2x[ r0 * width + c0 ] = left.x;
            result_2y[ r0 * width + c0 ] = left.y; 
        }
        else if (square_case == 4){
            result_1x[ r0 * width + c0 ] = left.x;
            result_1y[ r0 * width + c0 ] = left.y;
            result_2x[ r0 * width + c0 ] = bottom.x;
            result_2y[ r0 * width + c0 ] = bottom.y; 
        }
        else if (square_case == 5){
            result_1x[ r0 * width + c0 ] = top.x;
            result_1y[ r0 * width + c0 ] = top.y;
            result_2x[ r0 * width + c0 ] = bottom.x;
            result_2y[ r0 * width + c0 ] = bottom.y; 
        }
        else if (square_case == 6){
            // TODO !!!
            // result_1x[ r0 * width + c0 ] = 0.0;
            // result_1y[ r0 * width + c0 ] = 0.0;
            // result_2x[ r0 * width + c0 ] = 0.0;
            // result_2y[ r0 * width + c0 ] = 0.0; 
            result_1x[ r0 * width + c0 ] = left.x;
            result_1y[ r0 * width + c0 ] = left.y;
            result_2x[ r0 * width + c0 ] = top.x;
            result_2y[ r0 * width + c0 ] = top.y; 

        }
        else if (square_case == 7){
            result_1x[ r0 * width + c0 ] = right.x;
            result_1y[ r0 * width + c0 ] = right.y;
            result_2x[ r0 * width + c0 ] = bottom.x;
            result_2y[ r0 * width + c0 ] = bottom.y; 
        }
        else if (square_case == 8){
            result_1x[ r0 * width + c0 ] = bottom.x;
            result_1y[ r0 * width + c0 ] = bottom.y;
            result_2x[ r0 * width + c0 ] = right.x;
            result_2y[ r0 * width + c0 ] = right.y; 
        }
        else if (square_case == 9){
            // TODO !!!
            // result_1x[ r0 * width + c0 ] = 0.0;
            // result_1y[ r0 * width + c0 ] = 0.0;
            // result_2x[ r0 * width + c0 ] = 0.0;
            // result_2y[ r0 * width + c0 ] = 0.0; 
            result_1x[ r0 * width + c0 ] = top.x;
            result_1y[ r0 * width + c0 ] = top.y;
            result_2x[ r0 * width + c0 ] = left.x;
            result_2y[ r0 * width + c0 ] = left.y; 
        }
        else if (square_case == 10){
            result_1x[ r0 * width + c0 ] = bottom.x;
            result_1y[ r0 * width + c0 ] = bottom.y;
            result_2x[ r0 * width + c0 ] = top.x;
            result_2y[ r0 * width + c0 ] = top.y; 
        }
        else if (square_case == 11){
            result_1x[ r0 * width + c0 ] = bottom.x;
            result_1y[ r0 * width + c0 ] = bottom.y;
            result_2x[ r0 * width + c0 ] = left.x;
            result_2y[ r0 * width + c0 ] = left.y; 
        }
        else if (square_case == 12){
            result_1x[ r0 * width + c0 ] = left.x;
            result_1y[ r0 * width + c0 ] = left.y;
            result_2x[ r0 * width + c0 ] = right.x;
            result_2y[ r0 * width + c0 ] = right.y; 
        }
        else if (square_case == 13){
            result_1x[ r0 * width + c0 ] = top.x;
            result_1y[ r0 * width + c0 ] = top.y;
            result_2x[ r0 * width + c0 ] = right.x;
            result_2y[ r0 * width + c0 ] = right.y; 
        }
        else if (square_case == 14){
            result_1x[ r0 * width + c0 ] = left.x;
            result_1y[ r0 * width + c0 ] = left.y;
            result_2x[ r0 * width + c0 ] = top.x;
            result_2y[ r0 * width + c0 ] = top.y; 
        }
               
    } 

}
