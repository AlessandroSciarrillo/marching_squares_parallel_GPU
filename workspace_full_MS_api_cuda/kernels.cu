
#include <hip/hip_runtime.h>
extern "C" __device__
float get_fraction(double from_value, double to_value, double level){
    if (to_value == from_value)
        return 0;
    return ((level - from_value) / (to_value - from_value));
}

extern "C" __global__
void required_memory(double *image, size_t *result_required_memory, double level, size_t n, size_t width, size_t height)
{  
    size_t square_case;
    size_t r0 = blockIdx.y * blockDim.y + threadIdx.y;
    size_t c0 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t r1 = r0 + 1;
    size_t c1 = c0 + 1;
    
    if( r0 < height-1 && c0 < width-1 ){ 

        double ul = image[ r0 * width + c0 ]; 
        double ur = image[ r0 * width + c1 ];
        double ll = image[ r1 * width + c0 ];
        double lr = image[ r1 * width + c1 ];

        width = width - 1;

        square_case = 0;
        if (ul > level) square_case += 1;
        if (ur > level) square_case += 2;
        if (ll > level) square_case += 4;
        if (lr > level) square_case += 8; 

        if (square_case == 0 || square_case == 15){
            // 0
            result_required_memory[ r0 * width + c0 ] = 0;
        }
        else if (square_case == 6 || square_case == 9){
            // 2
            result_required_memory[ r0 * width + c0 ] = 2;
        }
        else {
            // 1
            result_required_memory[ r0 * width + c0 ] = 1;
        }    
    } 
}

extern "C" __global__
void reduce(size_t *required_memory, size_t *result_reduce, size_t n)
{ 
    __shared__ size_t temp[32]; // BLKDIM=32
    const size_t lindex = threadIdx.x;
    const size_t bindex = blockIdx.x;
    const size_t gindex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t bsize =  blockDim.x / 2;
    temp[lindex] = required_memory[gindex];
    __syncthreads();

    while( bsize > 0 ){
        if( lindex < bsize && (lindex+bsize)<n ){
            temp[lindex] += temp[lindex+bsize];
        }
        bsize = bsize / 2;
        __syncthreads();
    }
    if(0==lindex){
        result_reduce[bindex] = temp[0];
    }
}


/*
* Exclusive Scan
* Source. https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
*/
extern "C" __global__
void prescan(size_t *required_memory, size_t *result_exclusive_scan, size_t n) 
{ 
    __shared__ size_t temp[32]; // BLKDIM=32 // allocated on invocation 
    //size_t thid = threadIdx.x; 

    const size_t lindex = threadIdx.x;
    const size_t bindex = blockIdx.x;
    const size_t gindex = blockIdx.x * blockDim.x + threadIdx.x;

    size_t offset = 1; 
    temp[2*lindex] = required_memory[2*gindex]; // load input into shared memory 
    temp[2*lindex+1] = required_memory[2*gindex+1]; 
 	
    for (size_t d = n>>1; d > 0; d >>= 1) // build sum in place up the tree 
    { 
        __syncthreads();    
        if (lindex < d) { 
            size_t ai = offset*(2*lindex+1)-1;     
            size_t bi = offset*(2*lindex+2)-1;  
            temp[bi] += temp[ai];    
        }    
        offset *= 2; 
    } 

    if (lindex == 0) { temp[lindex - 1] = 0; } // clear the last element  
 	
    for (size_t d = 1; d < n; d *= 2){ // traverse down tree & build scan      
        offset >>= 1;      
        __syncthreads();      
        if (lindex < d) { 
            size_t ai = offset*(2*lindex+1)-1;     
            size_t bi = offset*(2*lindex+2)-1; 
 	
            size_t t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;       
        } 
    }  
    __syncthreads(); 

    result_exclusive_scan[2*gindex] = temp[2*lindex]; // write results to device memory      
    result_exclusive_scan[2*gindex+1] = temp[2*lindex+1]; 
} 